
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int *b, int* c){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(i<256){
        c[i] = a[i] + b[i];
    }
}

__managed__ int vector_a[256], vector_b[256], vector_c[256];



int main(){
    printf("Start...\n");

    for(int i=0; i<256; i++){
        vector_a[i] = i;
        vector_b[i] = 256-i;
    }

    add<<<1, 256 >>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();


    int result_sum = 0;

    for(int i=0; i<256; i++){
        result_sum += vector_c[i];
    }

    printf("End... Result: %d\n",result_sum);

    return 0;
}