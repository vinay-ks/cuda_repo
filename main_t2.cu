#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int* a, int* b, int* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // if (i < 256) {
        c[i] = a[i] + b[i];
    // }
}

int main() {
    printf("Devara Red Sea...\n");

    int *vector_a, *vector_b, *vector_c;

    // Allocate memory on the device (GPU)
    hipMalloc(&vector_a, 256 * sizeof(int));
    hipMalloc(&vector_b, 256 * sizeof(int));
    hipMalloc(&vector_c, 256 * sizeof(int));

    // Allocate temporary arrays on host
    int h_vector_a[256], h_vector_b[256], h_vector_c[256];

    // Initialize vectors on host
    for (int i = 0; i < 256; i++) {
        h_vector_a[i] = i;
        h_vector_b[i] = 256 - i;
    }

    // Copy vectors from host to device
    hipMemcpy(vector_a, h_vector_a, 256 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vector_b, h_vector_b, 256 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with 256 threads
    add<<<1, 256>>>(vector_a, vector_b, vector_c);

    // Synchronize device
    hipDeviceSynchronize();

    // Copy result back from device to host
    hipMemcpy(h_vector_c, vector_c, 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Compute the result sum on the host
    int result_sum = 0;
    for (int i = 0; i < 256; i++) {
        result_sum += h_vector_c[i];
    }

    printf("Done... Result: %d\n", result_sum);

    // Free device memory
    hipFree(vector_a);
    hipFree(vector_b);
    hipFree(vector_c);

    return 0;
}
